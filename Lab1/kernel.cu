#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void print_global_id()
{
    int globalId = (blockIdx.y * blockDim.y + threadIdx.y) * (gridDim.x * blockDim.x) + blockIdx.x * blockDim.x + threadIdx.x;
    printf("[DEVICE] GlobalId: %d\n", globalId);
}

int main()
{
    dim3 blockSize(4, 3, 1);
    dim3 gridSize(3, 3, 1);
    int* c_cpu;
    int* a_cpu;
    int* b_cpu;

    int* c_device;
    int* a_device;
    int* b_device;

    const int data_count = 10000;
    const int data_size = data_count * sizeof(int);

    c_cpu = (int*)malloc(data_size);
    a_cpu = (int*)malloc(data_size);
    b_cpu = (int*)malloc(data_size);

    hipMalloc((void**)&c_device, data_size);
    hipMalloc((void**)&a_device, data_size);
    hipMalloc((void**)&b_device, data_size);

    hipMemcpy(c_device, c_cpu, data_size, hipMemcpyHostToDevice);
    hipMemcpy(a_device, a_cpu, data_size, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_cpu, data_size, hipMemcpyHostToDevice);

    print_global_id << <gridSize, blockSize >> > ();

    hipMemcpy(c_cpu, c_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(a_cpu, a_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(b_cpu, b_device, data_size, hipMemcpyDeviceToHost);

    hipDeviceReset();
    hipFree(c_device);
    hipFree(a_device);
    hipFree(b_device);
    return 0;
}