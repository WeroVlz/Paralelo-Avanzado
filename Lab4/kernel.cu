#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define GPUErrorAssertion(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void convolution2D(int *matr, int *res, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < width && row < height) {
        int g_index = row * width + col;
        float sum = 0.0f;
        for (int i = -1; i <= 1; ++i) {
            for (int j = -1; j <= 1; ++j) {
                int curRow = row + i;
                int curCol = col + j;
                if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width) {
                    sum += matr[curRow * width + curCol];
                }
            }
        }
        res[g_index] = sum;
    }
}

int main() {
    const int width = 1024;
    const int height = 1024;
    const int dataSize = width * height * sizeof(int);

    int *MAT, *RES;
    int *MAT_gpu, *RES_gpu;

    MAT = (int*)malloc(dataSize);
    RES = (int*)malloc(dataSize);

    GPUErrorAssertion(hipMalloc((void**)&MAT_gpu, dataSize));
    GPUErrorAssertion(hipMalloc((void**)&RES_gpu, dataSize));

    for (int i = 0; i < width * height; ++i) {
        MAT[i] = rand() % 9;
    }

    printf("Before: \n");
    for (int i = 0; i < 15; ++i) {
        printf("MAT[%d] = %d\n", i, MAT[i]);

    }

    GPUErrorAssertion(hipMemcpy(MAT_gpu, MAT, dataSize, hipMemcpyHostToDevice));

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    convolution2D<<<gridSize, blockSize>>>(MAT_gpu, RES_gpu, width, height);
    GPUErrorAssertion(hipDeviceSynchronize());

    GPUErrorAssertion(hipMemcpy(RES, RES_gpu, dataSize, hipMemcpyDeviceToHost));

    printf("After: \n");
    for (int i = 0; i < 15; ++i) {
        printf("RES[%d] = %d\n", i, RES[i]);
    }

    hipFree(MAT_gpu);
    hipFree(RES_gpu);
    free(MAT);
    free(RES);

    return 0;
}
